#include "hip/hip_runtime.h"
// CUDA implementation using constant memory / constant memory + GL interop

#include "cuda_utils.h"
#include "Renderer.h"

static __constant__ Raycaster raycaster;
static __constant__ float4 transfer_fn[TF_SIZE];
static __constant__ esl_type esl_volume[ESL_VOLUME_SIZE];

GPURenderer2::GPURenderer2(Raycaster r) {
	set_window_buffer(r.view);
	set_transfer_fn(r);
	set_volume(r.volume);
}

GPURenderer3::GPURenderer3(Raycaster r) {
	set_window_buffer(r.view);
	set_transfer_fn(r);
	set_volume(r.volume);
}

static __global__ void render_ray(uchar4 dev_buffer[]) {
	short2 pos = {blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y};
	if ((pos.x >= raycaster.view.dims.x) || (pos.y >= raycaster.view.dims.y))	// ak su rozmery okna nedelitelne 16, spustaju sa prazdne thready
		return;

	float3 origin, direction;
	float2 k_range;
	raycaster.view.get_ray(pos, &origin, &direction); 
	if (!raycaster.intersect(origin, direction, &k_range))
		return;
	float3 pt = origin + (direction * k_range.x);
	while(k_range.x <= k_range.y) { 
		if (raycaster.esl && raycaster.sample_data_esl(esl_volume, pt)) 
			raycaster.leap_empty_space(pt, direction, &k_range);
		else 
			break;
		k_range.x += raycaster.ray_step;
		pt = origin + (direction * k_range.x);
	}
	if (k_range.x > k_range.y) 
		return;
	float4 color_acc = {0, 0, 0, 0};
	while (k_range.x <= k_range.y) {
		unsigned char sample = raycaster.volume.sample_data(pt);
		float4 color_cur = transfer_fn[sample / TF_RATIO];
		color_acc = color_acc + (color_cur * (1 - color_acc.w)); // transparency formula: C_out = C_in + C * (1-alpha_in); alpha_out = aplha_in + alpha * (1-alpha_in)
		if (color_acc.w > raycaster.ray_threshold) 
			break;
		k_range.x += raycaster.ray_step;
		pt = origin + (direction * k_range.x);
	}
	raycaster.write_color(color_acc, pos, dev_buffer);
}

void GPURenderer2::set_transfer_fn(Raycaster r) {
	cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(transfer_fn), r.transfer_fn, TF_SIZE * sizeof(float4)));
	cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(esl_volume), r.esl_volume, ESL_VOLUME_SIZE * sizeof(esl_type)));
}

int GPURenderer2::render_volume(uchar4 *buffer, Raycaster r) {
	r.volume.data = dev_volume_data;
	cuda_safe_call(hipMemset(dev_buffer, 0, dev_buffer_size));
	cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(raycaster), &r, sizeof(Raycaster)));
	render_ray<<<num_blocks, THREADS_PER_BLOCK>>>(dev_buffer);
	cuda_safe_check();
	cuda_safe_call(hipMemcpy(buffer, dev_buffer, dev_buffer_size, hipMemcpyDeviceToHost));
	return 0;
}

int GPURenderer3::render_volume(uchar4 *buffer, Raycaster r) {
	r.volume.data = dev_volume_data;
	cuda_safe_call(hipMemset(buffer, 0, dev_buffer_size));
	cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(raycaster), &r, sizeof(Raycaster)));
	render_ray<<<num_blocks, THREADS_PER_BLOCK>>>(buffer);
	cuda_safe_check();
	return 0;
}