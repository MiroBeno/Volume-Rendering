#include "hip/hip_runtime.h"
// Standard CUDA implementation

#include "data_utils.h"
#include "projection.h"
#include "model.h"

const int BUFFER_SIZE_CUDA = WIN_WIDTH * WIN_HEIGHT * 4;

static Volume_model volume;
static unsigned char *dev_buffer;

static hipEvent_t start, stop; 
static float elapsedTime;

__global__ void render_ray_gpu(Volume_model volume, Ortho_view view, unsigned char dev_buffer[]) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if ((col >= view.size_px.x) || (row >= view.size_px.y))					// ak su rozmery okna nedelitelne 16, spustaju sa prazdne thready
		return;

	float bg = (((col / 16) + (row / 16)) % 2) * 0.1f;
	float4 bg_color = {bg, bg, bg, 1};
	float4 color_acc;

	float3 origin = {0,0,0}, direction = {0,0,0};
	view.get_view_ray(col, row, &origin, &direction);
	float2 k_range = volume.intersect(origin, direction);

	if ((k_range.x < k_range.y) && (k_range.y > 0)) {				// nenulovy interval koeficientu k (existuje priesecnica) A vystupny bod lezi na luci
		if ((k_range.x < 0))										// bod vzniku luca je vnutri kocky, zaciname nie vstupnym priesecnikom, ale bodom vzniku
			k_range.x = 0;
		color_acc = make_float4(0,0,0,0);
		for (float k = k_range.x; k <= k_range.y; k += volume.ray_step) {		
			float3 pt = origin + (direction * k);
			float4 color_cur = volume.sample_color(pt);
			color_cur.x *= color_cur.w;								// transparency formula: C_out = C_in + C * (1-alpha_in); alpha_out = aplha_in + alpha * (1-alpha_in)
			color_cur.y *= color_cur.w;
			color_cur.z *= color_cur.w;
			color_acc = color_acc + (color_cur * (1 - color_acc.w));
			if (color_acc.w > 0.95f) 
				break;
		}
		color_acc = color_acc + (bg_color * (1 - color_acc.w));	
	}
	else {
		color_acc = bg_color;
	}

	int offset = (row * WIN_WIDTH + col) * 4;
	dev_buffer[offset + 0] = map_float_int(color_acc.x,256);
	dev_buffer[offset + 1] = map_float_int(color_acc.y,256);
	dev_buffer[offset + 2] = map_float_int(color_acc.z,256);
	dev_buffer[offset + 3] = 255;
}

extern void init_gpu(Volume_model volume_model) {
	volume = volume_model;
	unsigned char *dev_volume_data;
	hipMalloc((void **)&dev_volume_data, volume.size);
	hipMemcpy(dev_volume_data, volume.data, volume.size, hipMemcpyHostToDevice);
	volume.data = dev_volume_data;
	hipMalloc((void **)&dev_buffer, BUFFER_SIZE_CUDA);
	hipEventCreate(&start);
	hipEventCreate(&stop);
}

extern void free_gpu(void) {
	hipFree(dev_buffer);
	hipFree(volume.data);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

extern float render_volume_gpu(unsigned char *buffer, Ortho_view ortho_view) {
	int threads_dim = 16;
	dim3 threads_per_block(threads_dim, threads_dim);				// podla occupancy calculator
	dim3 num_blocks((WIN_WIDTH + threads_dim - 1) / threads_dim, (WIN_HEIGHT + threads_dim - 1) / threads_dim);		// celociselne delenie, 
																													// ak su rozmery okna nedelitelne 16, spustaju sa bloky	s nevyuzitimi threadmi
	hipEventRecord(start, 0);
	render_ray_gpu<<<num_blocks, threads_per_block>>>(volume, ortho_view, dev_buffer);
	hipMemcpy(buffer, dev_buffer, BUFFER_SIZE_CUDA, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	return elapsedTime;
}