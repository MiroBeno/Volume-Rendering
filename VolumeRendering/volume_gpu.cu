#include "hip/hip_runtime.h"
// Standard CUDA implementation

#include "Renderer.h"

#include "hip/hip_runtime_api.h"

uchar4 *GPURenderer::dev_buffer = NULL;
int GPURenderer::dev_buffer_size = 0;
unsigned char *GPURenderer::dev_volume_data = NULL;
dim3 GPURenderer::THREADS_PER_BLOCK(16, 16);				// pocet threadov v bloku - podla occupancy calculator
dim3 GPURenderer::num_blocks(0, 0);

GPURenderer1::GPURenderer1() {
	dev_transfer_fn = NULL;
}

GPURenderer1::~GPURenderer1() {
	hipFree(dev_buffer);
	hipFree(dev_volume_data);
	hipFree(dev_transfer_fn);
}

__global__ void render_ray(Raycaster raycaster, uchar4 dev_buffer[], unsigned char dev_volume_data[], float4 dev_transfer_fn[]) {
	int2 pos = {blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y};
	if ((pos.x >= raycaster.view.size_px.x) || (pos.y >= raycaster.view.size_px.y))	// ak su rozmery okna nedelitelne 16, spustaju sa prazdne thready
		return;

	float4 color_acc = {0,0,0,0};
	float3 origin, direction;
	float2 k_range;
	raycaster.view.get_ray(pos, &origin, &direction); 

	if (raycaster.intersect(origin, direction, &k_range)) {				
		for (float k = k_range.x; k <= k_range.y; k += raycaster.ray_step) {		
			float3 pt = origin + (direction * k);
			float4 color_cur = raycaster.sample_color(dev_volume_data, dev_transfer_fn, pt);
			color_acc = color_acc + (color_cur * (1 - color_acc.w)); // transparency formula: C_out = C_in + C * (1-alpha_in); alpha_out = aplha_in + alpha * (1-alpha_in)
			if (color_acc.w > raycaster.ray_threshold) 
				break;
		}
	}
	raycaster.write_color(color_acc, pos, dev_buffer);
}

void GPURenderer1::set_transfer_fn(float4 *transfer_fn) {
	if (dev_transfer_fn == NULL)
		hipMalloc((void **)&dev_transfer_fn, 256 * sizeof(float4));
	hipMemcpy(dev_transfer_fn, transfer_fn, 256 * sizeof(float4), hipMemcpyHostToDevice);
}

void GPURenderer1::set_window_size(int2 window_size) {
	if (dev_buffer != NULL)
		hipFree(dev_buffer);
	dev_buffer_size = window_size.x * window_size.y * 4;
	hipMalloc((void **)&dev_buffer, dev_buffer_size);
	num_blocks = dim3((window_size.x + THREADS_PER_BLOCK.x - 1) / THREADS_PER_BLOCK.x, 
					  (window_size.y + THREADS_PER_BLOCK.y - 1) / THREADS_PER_BLOCK.y);		
			// celociselne delenie, ak su rozmery okna nedelitelne 16, spustaju sa bloky s nevyuzitimi threadmi
}

void GPURenderer1::set_volume(Volume_model volume) {
	if (dev_volume_data != NULL)
		hipFree(dev_volume_data);
	hipMalloc((void **)&dev_volume_data, volume.size);
	hipMemcpy(dev_volume_data, volume.data, volume.size, hipMemcpyHostToDevice);
}

float GPURenderer1::render_volume(uchar4 *buffer, Raycaster *current_raycaster) {
	render_ray<<<num_blocks, THREADS_PER_BLOCK>>>(*current_raycaster, dev_buffer, dev_volume_data, dev_transfer_fn);
	hipMemcpy(buffer, dev_buffer, dev_buffer_size, hipMemcpyDeviceToHost);
	return 0;
}