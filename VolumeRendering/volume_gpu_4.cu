#include "hip/hip_runtime.h"
// CUDA implementation using constant memory + 3D texture memory + GL interop

#include "Renderer.h"

#include "hip/hip_runtime_api.h"
#include "hip/driver_types.h"
#include "driver_functions.h"
#include "hip/channel_descriptor.h"
#include "hip/hip_texture_types.h"
#include "texture_types.h"
#include ""

static __constant__ Raycaster raycaster;
//static __constant__ float4 transfer_fn[256];

hipArray *volume_array = 0;
texture<unsigned char, 3, hipReadModeNormalizedFloat> volume_texture;
hipArray *transfer_fn_array = 0;
texture<float4, 1, hipReadModeElementType> transfer_fn_texture;

GPURenderer4::GPURenderer4(int2 size, float4 *tf, Volume_model volume) {
	set_window_buffer(size);
	set_transfer_fn(tf);
	set_volume(volume);
}

GPURenderer4::~GPURenderer4() {
	hipUnbindTexture(volume_texture);
	hipFreeArray(volume_array);
	hipUnbindTexture(transfer_fn_texture);
	hipFreeArray(transfer_fn_array);
}

__device__ float4 sample_color_texture(float3 pos) {
	float sample = tex3D(volume_texture, (pos.x + 1)*0.5f, (pos.y + 1)*0.5f, (pos.z + 1)*0.5f);
	float4 color = tex1D(transfer_fn_texture, sample);
	//float4 color = transfer_fn[int(sample*255)]; 
	color.x *= color.w;				// aplikovanie optickeho modelu pre kompoziciu (farba * alfa)
	color.y *= color.w;
	color.z *= color.w;
	return color;
}

__global__ void render_ray(uchar4 dev_buffer[]) {
	int2 pos = {blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y};
	if ((pos.x >= raycaster.view.size_px.x) || (pos.y >= raycaster.view.size_px.y))	// ak su rozmery okna nedelitelne 16, spustaju sa prazdne thready
		return;

	float4 color_acc = {0,0,0,0};
	float3 origin, direction;
	float2 k_range;
	raycaster.view.get_ray(pos, &origin, &direction); 

	if (raycaster.intersect(origin, direction, &k_range)) {			
		for (float k = k_range.x; k <= k_range.y; k += raycaster.ray_step) {		
			float3 pt = origin + (direction * k);
			float4 color_cur = sample_color_texture(pt);
			color_acc = color_acc + (color_cur * (1 - color_acc.w)); // transparency formula: C_out = C_in + C * (1-alpha_in); alpha_out = aplha_in + alpha * (1-alpha_in)
			if (color_acc.w > raycaster.ray_threshold) 
				break;
		}
	}
	raycaster.write_color(color_acc, pos, dev_buffer);
}

void GPURenderer4::set_transfer_fn(float4 *tf) {
	if (transfer_fn_array == 0) {
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
		hipMallocArray(&transfer_fn_array, &channelDesc, 256, 1); 
		hipMemcpyToArray(transfer_fn_array, 0, 0, tf, 256 * sizeof(float4), hipMemcpyHostToDevice);

		transfer_fn_texture.filterMode = hipFilterModeLinear; //vypnut pri cm 
		transfer_fn_texture.normalized = true;
		transfer_fn_texture.addressMode[0] = hipAddressModeClamp; 
		hipBindTextureToArray(transfer_fn_texture, transfer_fn_array, channelDesc);
	}
	else {
		hipMemcpyToArray(transfer_fn_array, 0, 0, tf, 256 * sizeof(float4), hipMemcpyHostToDevice);
		//hipMemcpyToSymbol(HIP_SYMBOL(transfer_fn), tf, 256 * sizeof(float4));
	}
}

void GPURenderer4::set_volume(Volume_model volume) {
	hipExtent volumeDims = {volume.dims.x, volume.dims.y, volume.dims.z};	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();	
	hipMalloc3DArray(&volume_array, &channelDesc, volumeDims);

    hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr(dev_volume_data, volumeDims.width*sizeof(unsigned char), volumeDims.width, volumeDims.height);
    copyParams.dstArray = volume_array;
    copyParams.extent   = volumeDims;
    copyParams.kind     = hipMemcpyDeviceToDevice;				//!!z hosta
    hipMemcpy3D(&copyParams);

    volume_texture.normalized = true;                      
    volume_texture.filterMode = hipFilterModeLinear; //vypnut pri cm     
    volume_texture.addressMode[0] = hipAddressModeClamp;  
    volume_texture.addressMode[1] = hipAddressModeClamp;
    volume_texture.addressMode[2] = hipAddressModeClamp;
    hipBindTextureToArray(volume_texture, volume_array, channelDesc);
}

int GPURenderer4::render_volume(uchar4 *buffer, Raycaster *r) {
	hipMemcpyToSymbol(HIP_SYMBOL(raycaster), r, sizeof(Raycaster));
	render_ray<<<num_blocks, THREADS_PER_BLOCK>>>(buffer);
	return 0;
}