#include "hip/hip_runtime.h"
// CUDA implementation using constant memory

#include "data_utils.h"
#include "projection.h"
#include "model.h"
#include "raycaster.h"

#include "hip/hip_runtime_api.h"

extern dim3 THREADS_PER_BLOCK;

static __constant__ Raycaster raycaster;

extern unsigned char *dev_volume_data;
extern uchar4 *dev_buffer;
extern int dev_buffer_size;
extern hipEvent_t start, stop; 
extern float elapsedTime;

__global__ void render_ray_gpu2(uchar4 dev_buffer[]) {
	int2 pos = {blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y};
	if ((pos.x >= raycaster.view.size_px.x) || (pos.y >= raycaster.view.size_px.y))	// ak su rozmery okna nedelitelne 16, spustaju sa prazdne thready
		return;

	float4 color_acc = {0,0,0,0};
	float3 origin, direction;
	raycaster.view.get_ray(pos, &origin, &direction);
	float2 k_range = raycaster.intersect(origin, direction);

	if ((k_range.x < k_range.y) && (k_range.y > 0)) {				// nenulovy interval koeficientu k (existuje priesecnica) A vystupny bod lezi na luci
		for (float k = k_range.x; k <= k_range.y; k += raycaster.ray_step) {		
			float3 pt = origin + (direction * k);
			float4 color_cur = raycaster.sample_color(pt);
			color_acc = color_acc + (color_cur * (1 - color_acc.w)); // transparency formula: C_out = C_in + C * (1-alpha_in); alpha_out = aplha_in + alpha * (1-alpha_in)
			if (color_acc.w > raycaster.ray_threshold) 
				break;
		}
	}
	color_acc = color_acc + (raycaster.bg_color * (1 - color_acc.w));
	raycaster.write_color(color_acc, pos, dev_buffer);
}

extern float render_volume_gpu2(uchar4 *buffer, Raycaster current_raycaster) {
	current_raycaster.model.data = dev_volume_data;
	dim3 num_blocks((current_raycaster.view.size_px.x + THREADS_PER_BLOCK.x - 1) / THREADS_PER_BLOCK.x, 
					(current_raycaster.view.size_px.y + THREADS_PER_BLOCK.y - 1) / THREADS_PER_BLOCK.y);		
			// celociselne delenie, ak su rozmery okna nedelitelne 16, spustaju sa bloky s nevyuzitimi threadmi
	hipEventRecord(start, 0);
	hipMemcpyToSymbol(HIP_SYMBOL(raycaster), &current_raycaster, sizeof(Raycaster));
	render_ray_gpu2<<<num_blocks, THREADS_PER_BLOCK>>>(dev_buffer);
	hipMemcpy(buffer, dev_buffer, dev_buffer_size, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	return elapsedTime;
}