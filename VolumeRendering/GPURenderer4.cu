#include "hip/hip_runtime.h"
// CUDA implementation using constant memory + 3D texture memory + GL interop

#include "cuda_utils.h"
#include "Renderer.h"

static __constant__ Raycaster raycaster;
static __constant__ float4 transfer_fn[TF_SIZE];
//static __constant__ esl_type esl_volume[ESL_VOLUME_SIZE];

hipArray *volume_array = 0;
texture<unsigned char, 3, hipReadModeNormalizedFloat> volume_texture;
hipArray *transfer_fn_array = 0;
texture<float4, 1, hipReadModeElementType> transfer_fn_texture;
hipArray *esl_array = 0;
texture<esl_type, 2, hipReadModeElementType> esl_texture;

GPURenderer4::GPURenderer4(Raycaster r) {
	set_window_buffer(r.view);
	set_transfer_fn(r);
	set_volume(r.volume);
}

GPURenderer4::~GPURenderer4() {
	cuda_safe_call(hipUnbindTexture(volume_texture));
	cuda_safe_call(hipFreeArray(volume_array));
	cuda_safe_call(hipUnbindTexture(transfer_fn_texture));
	cuda_safe_call(hipFreeArray(transfer_fn_array));
	cuda_safe_call(hipUnbindTexture(esl_texture));
	cuda_safe_call(hipFreeArray(esl_array));
}

__device__ float4 sample_texture_interpolated(float sample) {
	float index = sample * (TF_SIZE-1);
	float floor_index = floor(index);
	float ceil_index = ceil(index);
	return floor_index != ceil_index
					? transfer_fn[(int) floor_index] * (ceil_index - index) + transfer_fn[(int) ceil_index] * (index - floor_index)
					: transfer_fn[(int) index];
}

__device__  bool sample_data_esl_texture(float3 pos) {
		esl_type sample = tex2D(esl_texture, 
							map_float_int((pos.y + 1)*0.5f, raycaster.volume.dims.y) / raycaster.esl_block_dims,
							map_float_int((pos.z + 1)*0.5f, raycaster.volume.dims.z) / raycaster.esl_block_dims);
		unsigned short index = map_float_int((pos.x + 1)*0.5f, raycaster.volume.dims.x) / raycaster.esl_block_dims;
		return ((sample & (1 << index)) != 0);
}

__device__ void shade_texture(float4 *color, float3 pos, float sample) {
		if (color->w < 0.1f || raycaster.light_kd < 0.01f) 
			return;
		float3 light_dir = vector_normalize(raycaster.view.light_pos - pos);
		float sample_l = tex3D(volume_texture, 
			(pos.x + light_dir.x * 0.01f + 1)*0.5f,
			(pos.y + light_dir.y * 0.01f + 1)*0.5f,
			(pos.z + light_dir.z * 0.01f + 1)*0.5f);
		float diffuse_light = (sample_l - sample) * raycaster.light_kd;
		color->x += diffuse_light;
		color->y += diffuse_light;
		color->z += diffuse_light;
}

static __global__ void render_ray(uchar4 dev_buffer[]) {
	short2 pos = {blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y};
	if ((pos.x >= raycaster.view.dims.x) || (pos.y >= raycaster.view.dims.y))	// ak su rozmery okna nedelitelne 16, spustaju sa prazdne thready
		return;

	float3 origin, direction;
	float2 k_range;
	raycaster.view.get_ray(pos, &origin, &direction); 
	if (!raycaster.intersect(origin, direction, &k_range)) 
		return;
	float3 pt = origin + (direction * k_range.x);
	while(k_range.x <= k_range.y) { 
		if (raycaster.esl && sample_data_esl_texture(pt)) 
			raycaster.leap_empty_space(pt, direction, &k_range);
		else 
			break;
		k_range.x += raycaster.ray_step;
		pt = origin + (direction * k_range.x);
	}
	if (k_range.x > k_range.y) 
		return;
	float4 color_acc = {0, 0, 0, 0};
	//color_acc = color_acc + (make_float4(0.5f, 0.5f, 1, 0.5f) * (1 - color_acc.w));
	while (k_range.x <= k_range.y) {
		float sample = tex3D(volume_texture, (pt.x + 1)*0.5f, (pt.y + 1)*0.5f, (pt.z + 1)*0.5f);
		float4 color_cur = tex1D(transfer_fn_texture, sample);
		//float4 color = transfer_fn[int(sample*(TF_SIZE-1))];
		shade_texture(&color_cur, pt, sample);
		color_acc = color_acc + (color_cur * (1 - color_acc.w)); // transparency formula: C_out = C_in + C * (1-alpha_in); alpha_out = aplha_in + alpha * (1-alpha_in)
		if (color_acc.w > raycaster.ray_threshold) 
			break;
		k_range.x += raycaster.ray_step;
		pt = origin + (direction * k_range.x);
	}
	raycaster.write_color(color_acc, pos, dev_buffer);
}

void GPURenderer4::set_transfer_fn(Raycaster r) {
	if (transfer_fn_array == 0) {
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
		cuda_safe_call(hipMallocArray(&transfer_fn_array, &channelDesc, TF_SIZE, 1)); 

		transfer_fn_texture.filterMode = hipFilterModeLinear; 
		transfer_fn_texture.normalized = true;
		transfer_fn_texture.addressMode[0] = hipAddressModeClamp; 
		cuda_safe_call(hipBindTextureToArray(transfer_fn_texture, transfer_fn_array, channelDesc));
	}
	cuda_safe_call(hipMemcpyToArray(transfer_fn_array, 0, 0, r.transfer_fn, TF_SIZE * sizeof(float4), hipMemcpyHostToDevice));
	/**/cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(transfer_fn), r.transfer_fn, TF_SIZE * sizeof(float4)));

	if (esl_array == 0) {
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<esl_type>();	
		cuda_safe_call(hipMallocArray(&esl_array, &channelDesc, ESL_VOLUME_DIMS, ESL_VOLUME_DIMS));

		esl_texture.normalized = false;
		esl_texture.filterMode = hipFilterModePoint;  
		esl_texture.addressMode[0] = hipAddressModeClamp;  
		esl_texture.addressMode[1] = hipAddressModeClamp;
		cuda_safe_call(hipBindTextureToArray(esl_texture, esl_array, channelDesc));
	}
	cuda_safe_call(hipMemcpyToArray(esl_array, 0, 0, r.esl_volume, ESL_VOLUME_SIZE * sizeof(esl_type), hipMemcpyHostToDevice));
	//cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(esl_volume), r.esl_volume, ESL_VOLUME_SIZE * sizeof(esl_type)));
}

void GPURenderer4::set_volume(Model volume) {
	hipExtent volumeDims = {volume.dims.x, volume.dims.y, volume.dims.z};	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();	
	cuda_safe_call(hipMalloc3DArray(&volume_array, &channelDesc, volumeDims));

    hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr(volume.data, volumeDims.width*sizeof(unsigned char), volumeDims.width, volumeDims.height);
    copyParams.dstArray = volume_array;
    copyParams.extent   = volumeDims;
    copyParams.kind     = hipMemcpyHostToDevice;
    cuda_safe_call(hipMemcpy3D(&copyParams));

    volume_texture.normalized = true;                      
    volume_texture.filterMode = hipFilterModeLinear; //vypnut pri cm ?   
    volume_texture.addressMode[0] = hipAddressModeClamp;  
    volume_texture.addressMode[1] = hipAddressModeClamp;
    volume_texture.addressMode[2] = hipAddressModeClamp;
    cuda_safe_call(hipBindTextureToArray(volume_texture, volume_array, channelDesc));
}

int GPURenderer4::render_volume(uchar4 *buffer, Raycaster r) {
	cuda_safe_call(hipMemset(buffer, 0, dev_buffer_size));
	cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(raycaster), &r, sizeof(Raycaster)));
	render_ray<<<num_blocks, THREADS_PER_BLOCK>>>(buffer);
	cuda_safe_check();
	return 0;
}